﻿#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(float* a, float* b, float* c, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n) c[i] = a[i] + b[i];
}

int main() {
	const int N = 1000;
	size_t size=N*sizeof(float);

	float* h_a, * h_b, * h_c;
	h_a = (float*)malloc(size);
	h_b = (float*)malloc(size);
	h_c = (float*)malloc(size);

	for (int i = 0; i < N;i++) {
		h_a[i] = 1.0f;
		h_b[i] = 2.0f;
	}

	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	add <<< blocksPerGrid, threadsPerBlock >>>(d_a, d_b, d_c, N);

	hipMemcpy(h_c,d_c, size, hipMemcpyDeviceToHost);
	printf("Result[0] = %f\n", h_c[0]);

	free(h_a);free(h_b);free(h_c);
	hipFree(d_a);hipFree(d_b);hipFree(d_c);
	return 0;
}